
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>

void printFloatArray(const float arr[], int size) {
  // std::cout << size;
  for (int i = 0; i < size; i++) {
    std::cout << arr[i];
    if (i < size - 1) {
      std::cout << ", ";
    }
  }
  std::cout << std::endl;
}

void vecadd(float *A_h, float *B_h, float *C_h, int N) {
  for (size_t i = 0; i < N; i++) {
    C_h[i] = A_h[i] + B_h[i];
  }
}

// The __global__ is a function identifier which says to CUDA that this is a
// kernal function and it can be called to generate a grid of threads on a
// device
__global__ void vec_add_kernal(float *A, float *B, float *C, int N) {
  /*
  blockDim : For a given grid of threads, the number of threads in a block is
  available in a built-in variable named blockDim .

  threadIdx : The threadIdx variable gives each thread a unique coordinate
  within a block. The first thread in each block has value 0 in its threadIdx.x
  variable, the second thread has value 1, the third thread has value 2, and so
  on.

  blockIdx : The blockIdx  variable gives all threads in a block a common block
  coordinate. All threads in the first block have value 0 in their blockIdx.x
  variables, those in the second thread block value 1, and so on.
  */
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < N) {
    C[i] = A[i] + B[i];
  }
}

void vecadd_gpu(float *A_h, float *B_h, float *C_h, int N) {
  int size = N * sizeof(float);
  float *A_d, *B_d, *C_d;

  // cudaMalloc is used to allocate memory on the device
  hipMalloc((void **)&A_d, size);
  hipMalloc((void **)&B_d, size);
  hipMalloc((void **)&C_d, size);

  // Function to copy data to and from device and host. Uses either
  // cudaMemcpyHostToDevice or cudaMemcpyDeviceToHost to inidicate direction
  hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
  hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

  // Mechanism to declare execution parameters more formally
  dim3 threads(256, 1, 1);
  dim3 blocks(ceil(N / 256.0), 1, 1);

  // The tripe quote thingy is an execution parameter
  vec_add_kernal<<<blocks, threads>>>(A_d, B_d, C_d, N);

  hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);

  // Use yer imagination
  // printFloatArray(C_d,N);
  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_d);
}

int main(int argc, char const *argv[]) {
  hipDeviceSynchronize();
  int N = argc > 1 ? atoi(argv[1]) : 100;
  // int N = atoi(argv[1]);
  float *A = new float[N];
  float *B = new float[N];
  float *C = new float[N];
  std::fill(A, A + N, 1.0 * 14);
  std::fill(B, B + N, 1.0 * 16);
  // printFloatArray(A, N);
  // printFloatArray(B, N);

  // vecadd(A, B, C, N);
  vecadd_gpu(A, B, C, N);
  printFloatArray(C, N);
  return 0;
}
// nvcc simple_add.cu -o simple_add_gpu & ./simple_add_gpu 10